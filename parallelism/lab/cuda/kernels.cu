#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat (float *dev_u, float *dev_uhelp, int N) {

  //extern __shared__ float sdata[]; // lifetime of the block

  //int tid = threadIdx.y*blockDim.x + threadIdx.x;

  int col = (blockIdx.x * blockDim.x) + threadIdx.x;
  int row = (blockIdx.y * blockDim.y) + threadIdx.y;

  //sdata[tid] = dev_u[row*N + col];
  //__syncthreads();

  if (row > 0 && row < (N-1) && col > 0 && col < (N-1)) {
    //dev_uhelp[row*N + col] = 0.25 * ( sdata[(threadIdx.y-1)*blockDim.x + threadIdx.x]
                                    //+ sdata[(threadIdx.y+1)*blockDim.x + threadIdx.x]
                                    //+ sdata[threadIdx.y*blockDim.x + (threadIdx.x - 1)]
                                    //+ sdata[threadIdx.y*blockDim.x + (threadIdx.x + 1)]
                                    //);
    dev_uhelp[row*N + col] = 0.25 * ( dev_u[row*N + (col-1)]
                                    + dev_u[row*N + (col+1)]
                                    + dev_u[(row-1)*N + col]
                                    + dev_u[(row+1)*N + col]
                                    );
  }
}

__global__ void gpu_Heat2 (float *dev_u, float *dev_uhelp, float *dev_r, int N) {

  int col = (blockIdx.x * blockDim.x) + threadIdx.x;
  int row = (blockIdx.y * blockDim.y) + threadIdx.y;

  float diff;

  if (row > 0 && row < (N-1) && col > 0 && col < (N-1)) {
    dev_uhelp[row*N + col] = 0.25 * ( dev_u[row*N + (col-1)]
                                    + dev_u[row*N + (col+1)]
                                    + dev_u[(row-1)*N + col]
                                    + dev_u[(row+1)*N + col]
                                    );
    diff = dev_uhelp[row*N + col] - dev_u[row*N + col];
    dev_r[row*N + col] = diff * diff;
  }
}
