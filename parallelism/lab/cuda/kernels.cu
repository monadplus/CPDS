#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat0 (float *dev_u, float *dev_uhelp, float *dev_r, int N) {

  int col = (blockIdx.x * blockDim.x) + threadIdx.x;
  int row = (blockIdx.y * blockDim.y) + threadIdx.y;

  float diff;

  if (row > 0 && row < (N-1) && col > 0 && col < (N-1)) {
    dev_uhelp[row*N + col] = 0.25 * ( dev_u[row*N + (col-1)]
                                    + dev_u[row*N + (col+1)]
                                    + dev_u[(row-1)*N + col]
                                    + dev_u[(row+1)*N + col]
                                    );
    diff = dev_uhelp[row*N + col] - dev_u[row*N + col];
    dev_r[row*N + col] = diff * diff;
  }
}

__global__ void gpu_Residuals0 (float *in, float *out, int N) {

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  if (i < N) {

    sdata[tid] = in[i];
    __syncthreads();

    for(unsigned int s=1; s < blockDim.x && i+s < N; s *= 2) {
      if (tid % (2*s) == 0) {
        sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
    }

    if (tid == 0) out[blockIdx.x] = sdata[0];
  }
}
