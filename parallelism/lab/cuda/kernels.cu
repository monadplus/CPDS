#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat0 (float *dev_u, float *dev_uhelp, float *dev_r, int N) {

  int col = (blockIdx.x * blockDim.x) + threadIdx.x;
  int row = (blockIdx.y * blockDim.y) + threadIdx.y;

  float diff = 0.0;

  if (row > 0 && row < N-1 && col > 0 && col < N-1) {
    dev_uhelp[row*N + col] = 0.25 * ( dev_u[row*N + (col-1)]
                                    + dev_u[row*N + (col+1)]
                                    + dev_u[(row-1)*N + col]
                                    + dev_u[(row+1)*N + col]
                                    );
    diff = dev_uhelp[row*N + col] - dev_u[row*N + col];
    dev_r[(row-1)*(N-2) + col - 1] = diff * diff;
  }
}

// =============================================================

// Check bounds: this version allows the last block to be half empty.
__global__ void gpu_Reduce0 (float *in, float *out, int N) {

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  if (i < N) {

    sdata[tid] = in[i];
    __syncthreads();

    for(unsigned int s=1; s < blockDim.x && i+s < N; s *= 2) {
      if (tid % (2*s) == 0) {
        sdata[tid] += sdata[tid + s];
      }
      __syncthreads();
    }

    if (tid == 0) out[blockIdx.x] = sdata[0];
  }
}

// =============================================================

__global__ void gpu_Reduce1 (float *in, float *out) {

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  sdata[tid] = in[i];
  __syncthreads();

  for(unsigned int s=1; s < blockDim.x; s *= 2) {
    if (tid % (2*s) == 0) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) out[blockIdx.x] = sdata[0];
}

// =============================================================

// Divergency in warps removed
// Remove % (slow)
__global__ void gpu_Reduce2 (float *in, float *out) {

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  sdata[tid] = in[i];
  __syncthreads();

  for(unsigned int s=1; s < blockDim.x; s *= 2) {
    int index = 2*s*tid;
    if (index < blockDim.x) {
      sdata[index] += sdata[index + s];
    }
    __syncthreads();
  }

  if (tid == 0) out[blockIdx.x] = sdata[0];
}

// =============================================================

// 32 banks of 64-bits (or 32-bits mode).
//
// if two addresses of a memory request fall in the same memory bank, there is a bank conflict and the access has to be serialized.
//
// A shared memory request for a warp does not generate a bank conflict between two threads that access any sub-word within the same 64-bit word (even though the addresses of the two sub-words fall in the same bank).
__global__ void gpu_Reduce3 (float *in, float *out) {

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  sdata[tid] = in[i];
  __syncthreads();

  for(unsigned int s=blockDim.x*0.5; s>0; s>>=1) {
    if (tid < s) {
      sdata[tid] += sdata[tid+s];
    }
    __syncthreads();
  }

  if (tid == 0) out[blockIdx.x] = sdata[0];
}

// =============================================================

// First add, half of the threads where idle.
// This version is the first one to match the cpu output.
// NOTE: you need to halve the #blocks on the call.
__global__ void gpu_Reduce4 (float *in, float *out) {

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

  // Each block processes two blocks
  sdata[tid] = in[i] + in[i+blockDim.x];
  __syncthreads();

  // Now we have half the blocks.
  for(unsigned int s=blockDim.x*0.5; s>0; s>>=1) {
    if (tid < s) {
      sdata[tid] += sdata[tid+s];
    }
    __syncthreads();
  }

  if (tid == 0) out[blockIdx.x] = sdata[0];
}

// =============================================================

// These optimizations can be disabled using the volatile keyword: If a variable located in global or shared memory is declared as volatile, the compiler assumes that its value can be changed or used at any time by another thread and therefore any reference to this variable compiles to an actual memory read or write instruction.
__device__ void warpReduce0(volatile float* sdata, int tid) {
  // Warps: between instructions there is an implicit barrier
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid +  8];
  sdata[tid] += sdata[tid +  4];
  sdata[tid] += sdata[tid +  2];
  sdata[tid] += sdata[tid +  1];
}

// Warps are synchronized 32 threads.
// Unroll last iterations where threads < 32
__global__ void gpu_Reduce5 (float *in, float *out) {

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

  sdata[tid] = in[i] + in[i+blockDim.x];
  __syncthreads();

  // Saves work:
  //   All warps execute every iteration of the loop
  //    and the if statement.
  for(unsigned int s=blockDim.x*0.5; s>32; s>>=1) {
    if (tid < s) {
      sdata[tid] += sdata[tid+s];
    }
    __syncthreads();
  }

  if (tid < 32) warpReduce0(sdata, tid);

  if (tid == 0) out[blockIdx.x] = sdata[0];
}

// =============================================================

// Loop unrolling: templates are evaluated at compile time.

template <unsigned int blockSize>
    __device__ void warpReduce1(volatile float* sdata, int tid) {
  if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
  if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
  if (blockSize >= 16) sdata[tid] += sdata[tid +  8];
  if (blockSize >= 8)  sdata[tid] += sdata[tid +  4];
  if (blockSize >= 4)  sdata[tid] += sdata[tid +  2];
  if (blockSize >= 2)  sdata[tid] += sdata[tid +  1];
}

template <unsigned int blockSize>
    __global__ void gpu_Reduce6 (float *in, float *out) {

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

  sdata[tid] = in[i] + in[i+blockDim.x];
  __syncthreads();

  if (blockSize >= 512) {
    if (tid < 256) {sdata[tid] += sdata[tid + 256];} __syncthreads(); }
  if (blockSize >= 256) {
    if (tid < 128) {sdata[tid] += sdata[tid + 128];} __syncthreads(); }
  if (blockSize >= 128) {
    if (tid < 64) {sdata[tid] += sdata[tid + 64];} __syncthreads(); }

  if (tid < 32) warpReduce1<blockSize>(sdata, tid);

  if (tid == 0) out[blockIdx.x] = sdata[0];
}

// =============================================================

// Multiple adds per thread

// NOTE: you need to reduce the number of blocks
template <unsigned int blockSize>
    __global__ void gpu_Reduce7 (float *in, float *out, unsigned int N) {

  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  unsigned int gridSize = blockSize*2*gridDim.x; // grid = #blocks
  sdata[tid] = 0;

  while (i < N) {
    sdata[tid] += in[i] + in[i+blockSize];
    i += gridSize;
  }

  __syncthreads();

  if (blockSize >= 512) {
    if (tid < 256) {sdata[tid] += sdata[tid + 256];} __syncthreads(); }
  if (blockSize >= 256) {
    if (tid < 128) {sdata[tid] += sdata[tid + 128];} __syncthreads(); }
  if (blockSize >= 128) {
    if (tid < 64) {sdata[tid] += sdata[tid + 64];} __syncthreads(); }

  if (tid < 32) warpReduce1<blockSize>(sdata, tid);

  if (tid == 0) out[blockIdx.x] = sdata[0];
}
